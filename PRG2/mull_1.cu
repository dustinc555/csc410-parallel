#include "hip/hip_runtime.h"
/** Thrust Library **/
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

/** Std library **/
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <random>
#include <time.h>
#include <chrono>


__global__ void performMults(double * a, double * b, int ROW_SIZE, int SIZE)
{
  int a_index = blockIdx.x * blockDim.x + threadIdx.x;

  int b_index = a_index % ROW_SIZE;

  if (a_index >= SIZE) return;
  // The multiplication stage must be done before the mapping and reduction stage
  // all of these tasks can be done in parallel
  a[a_index] *= b[b_index];

}


using namespace std;

/** sumRows(double * arr, double * b, double * c, const int N, const int SIZE)
*   Expects arr to be a matrix, and c a result vector
*   c[i] = sum(a[i,j] * b[i])
*  
*/
__global__ void sumRows(double * a, double * c, const int ROW_SIZE, const int SIZE)
{
  #ifndef TIMED
  int a_index = blockIdx.x * blockDim.x + threadIdx.x;
  #else
  int a_index = blockIdx.x;
  #endif

  int b_index = a_index % ROW_SIZE; // you can consider b_index the row id (0 start, ROW_SIZE-1 end)
  
  if (b_index == 0) // if we are a zero index, sum up the row up to but not including the next 0 row.
  {
    int local_c_sum = 0;
    for (int i = 0; i < ROW_SIZE; i++)
      local_c_sum += a[a_index + i];

    int c_index = a_index / ROW_SIZE;
    c[c_index] = local_c_sum; 
  }
  // this method is bad because its tasks size grow with the problem instead of the number of tasks. 
}

const int INCORRECT_NUM_ARGS_ERROR = 1;
const unsigned THREADS = 512;

void usage();


/**** MAIN ***********************/
/*********************************/
int main( int argc, char* argv[] )
{
  if ( argc < 3 )
    usage();
 
  unsigned threads = THREADS;
  const int N = atoi(argv[1]);
  const int SIZE = N * N; // square matrix N by N

  thrust::host_vector<double> h_a(SIZE);
  thrust::host_vector<double> h_b(N);
  thrust::device_vector<double> d_a(SIZE, 1);
  thrust::device_vector<double> d_b(N, 1);
  thrust::device_vector<double> c(N);
  
  #ifndef TIMED 
  bool random = argv[2][0] == 'r';
  #else
  bool random = argv[3][0] == 'r';
  threads = atoi(argv[2]);
  #endif

  double lowerLimit = random ? 0 : 1;
  double upperLimit = random ? 3 : 1;
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();

  #ifdef DEBUG
  printf("upperLimit: %f  lowerLimit: %f\n", upperLimit, lowerLimit);
  #endif

  std::default_random_engine re(seed);
  std::uniform_real_distribution<double> unif(lowerLimit,upperLimit);
  for (int i = 0; i < h_a.size(); i++)
    h_a[i] = floor(unif(re));
  for (int i = 0; i < h_b.size(); i++)
    h_b[i] = floor(unif(re));
  

  d_a = h_a;
  d_b = h_b;
  
  #ifdef DEBUG
  cout << "Matrix values:" << endl;
  for (int i = 0; i < SIZE; i++) 
  {
    cout << h_a[i] << " ";
    if ((i + 1) % N == 0) cout << endl;
  }
  cout << "\n\n";
  cout << "Vector values:" << endl;
  for (int i = 0; i < N; i++)
    cout << h_b[i] << " ";
  cout << endl;
  #endif

  // vectors are unfortunatly not available on cuda device
  // but you can get the memory address, pass it to the device,
  // and treat it as a normal array.
  double * p_a = thrust::raw_pointer_cast(&d_a[0]);
  double * p_b = thrust::raw_pointer_cast(&d_b[0]);
  double * p_c = thrust::raw_pointer_cast(&c[0]);

  unsigned blocks;
  // one thread per block, if doing the Karp-Flatt Metric
  #ifdef TIMED
  blocks = threads;
  threads = 1;
  #else
  // just make sure that there are enough threads
  blocks = (SIZE / threads) + 1;
  #endif


  // record action time 
  #ifdef TIMED
  auto start = chrono::steady_clock::now();
  #endif

  performMults<<<blocks, threads>>>(p_a, p_b, N, SIZE);
  hipDeviceSynchronize(); 
  sumRows<<<blocks, threads>>>(p_a, p_c, N, SIZE);
  hipDeviceSynchronize();

  #ifdef TIMED
  auto end = chrono::steady_clock::now();
  cout << chrono::duration_cast<chrono::nanoseconds>(end - start).count();
  #endif


  thrust::host_vector<double> result = c;

  #ifdef DEBUG
  printf("\n\nresult:\n");
  #endif

  #ifndef TIMED
  for (int i = 0; i < result.size(); i++)
    cout << result[i] << " ";
  #endif

  #ifdef DEBUG 
  cout << endl;
  #endif
   
  return 0;
} 


void usage()
{
  printf("./main <N> <mode>\n");
  printf("mode: 1 to fill matrix and vector with all 1's.\n");
  printf("\tr for all random numbers.\n");
  printf("if make Timed: ./main <N> <threads> <mode>\n");
  exit(INCORRECT_NUM_ARGS_ERROR);
}




